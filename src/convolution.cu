#include "hip/hip_runtime.h"
#include "../include/convolution.h"

#define PRINTCONVOLUTIONBENCH() PRINTCUDABENCH2(MS)

__global__ void mooreFilter2d(
    const unsigned char *srcimg,
    unsigned char *dstimg,
    const size_t width,
    const size_t height,
    const winType *window,
    const unsigned windowWidth,
    const unsigned windowSize,
    const unsigned windowElems,
    const int cn)
{
    unsigned x = blockIdx.x * blockDim.x + threadIdx.x,
             y = blockIdx.y * blockDim.y + threadIdx.y;
    const size_t index = width * y + x;
    const size_t indexcn = index * cn;

    unsigned ndiv2 = windowWidth / 2;
    if (x < width - ndiv2 && x > ndiv2 - 1 && y < height - ndiv2 && y > ndiv2 - 1)
    {
        unsigned sum0 = 0, sum1 = 0, sum2 = 0;
        const size_t winSizecn = windowSize * cn;
        const size_t winWidthcn = windowWidth * cn;

        size_t icn = 0, tmp = (index - (windowWidth / 2) * (width + 1)) * cn;

        while (icn < winSizecn)
        {
            for (size_t i = 0; i < windowWidth; ++i, icn += cn, tmp += cn)
            {
                // printf("(%u,%u) index: %lu, cn: %u, indexcn: %lu, icn: %lu, tmp: %lu, winSizecn: %lu\n", x, y, index, cn, indexcn, icn, tmp, winSizecn);
                sum0 += window[icn + 0] * srcimg[tmp + 0];
                sum1 += window[icn + 1] * srcimg[tmp + 1];
                sum2 += window[icn + 2] * srcimg[tmp + 2];
            }
            // printf("(%u,%u) loop1 out\n", x, y);
            tmp = tmp + (width * cn) - winWidthcn;
        }
        // printf("(%u,%u) loop2 out\n", x, y);

        dstimg[indexcn + 0] = sum0 / windowSize;
        dstimg[indexcn + 1] = sum1 / windowSize;
        dstimg[indexcn + 2] = sum2 / windowSize;
    }
    else
    {
        dstimg[indexcn + 0] = srcimg[indexcn + 0];
        dstimg[indexcn + 1] = srcimg[indexcn + 1];
        dstimg[indexcn + 2] = srcimg[indexcn + 2];
    }
}

void convolve2d(
    const cv::Mat &image,
    const Window &window,
    void (*convolveFunc2d)(const unsigned char *, unsigned char *, const size_t, const size_t, const winType *, const unsigned, const unsigned, const unsigned, const int),
    const int blockWidth,
    const int blockHeight)
{
    INITCUDADBG();

    const size_t numOfPixels = image.total();
    const int cn = image.channels();
    const size_t numOfElems = cn * numOfPixels;
    const int width = image.cols;
    const int height = image.rows;

    unsigned char *srcimg, *dstimg;
    winType *dwindow;

    CUDADBG(hipMalloc(&dstimg, numOfElems * sizeof(unsigned char)), );
    CUDADBG(hipMalloc(&srcimg, numOfElems * sizeof(unsigned char)), );
    CUDADBG(hipMemcpy(srcimg, image.data, numOfElems * sizeof(unsigned char), hipMemcpyHostToDevice), );

    CUDADBG(hipMalloc(&dwindow, window.elems * sizeof(winType)), );
    CUDADBG(hipMemcpy(dwindow, window.data.data(), window.elems * sizeof(winType), hipMemcpyHostToDevice), );

    const dim3 blockSize(blockWidth, blockHeight, 1);
    const dim3 gridSize((width - 1) / blockWidth + 1, (height - 1) / blockHeight + 1, 1);
    INITCUDABENCH();

    STARTCUDABENCH();
    convolveFunc2d<<<gridSize, blockSize>>>(srcimg, dstimg, width, height, dwindow, window.width, window.size, window.elems, cn);
    CUDACHECK();
    STOPCUDABENCH();
    PRINTCONVOLUTIONBENCH();

    CUDADBG(hipMemcpy(image.data, dstimg, numOfElems * sizeof(unsigned char), hipMemcpyDeviceToHost), );

    CUDADBG(hipFree(dstimg), );
    CUDADBG(hipFree(dwindow), );
}