#include "hip/hip_runtime.h"
#include "../include/gamma.h"

__global__ void gammaAvgKernel(
    unsigned char *dstimg,
    const int cn,
    const size_t width, const size_t height,
    const size_t numOfPixels,
    const float *gammaVals,
    const size_t howmany)
{
    unsigned x = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned y = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned i = width * y + x;
    unsigned icn = i * cn;

    if (i < numOfPixels)
    {
        float result = 0;
        for (size_t j = 0; j < howmany; ++j)
        {
            result += dstimg[icn + j] * gammaVals[j];
        }
        for (size_t j = 0; j < howmany; ++j)
        {
            dstimg[icn + j] = result;
        }
    }
}
__global__ void gammaKernel(
    unsigned char *dstimg,
    const int cn,
    const size_t width, const size_t height,
    const size_t numOfPixels,
    const float *gammaVals,
    const size_t howmany)
{
    unsigned x = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned y = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned i = width * y + x;
    unsigned icn = i * cn;

    if (i < numOfPixels)
    {
        for (size_t j = 0; j < howmany; ++j)
        {
            dstimg[icn + j] *= gammaVals[j];
        }
    }
}

void gammaFilter(
    const cv::Mat &image,
    std::vector<float> gammaVals, void (*gammaFunc)(unsigned char *, const int, const size_t, const size_t, const size_t, const float *, const size_t),
    const int blockWidth,
    const int blockHeight)
{
    INITCUDADBG();

    const size_t numOfPixels = image.total();
    const int cn = image.channels();
    const size_t numOfElems = cn * numOfPixels;
    const int width = image.cols;
    const int height = image.rows;

    size_t howmany = gammaVals.size();

    if (howmany > cn)
    {
        ERROR("gamma size greater than number of channels.", )
    }

    float *gammaValsPtr;
    CUDADBG(hipMalloc(&gammaValsPtr, numOfElems * sizeof(unsigned char) + howmany * sizeof(float)), );
    unsigned char *dstimg = (unsigned char *)(gammaValsPtr + howmany);

    CUDADBG(hipMemcpy(dstimg, image.data, numOfElems * sizeof(unsigned char), hipMemcpyHostToDevice), );
    CUDADBG(hipMemcpy(gammaValsPtr, gammaVals.data(), howmany * sizeof(float), hipMemcpyHostToDevice), );

    const dim3 blockSize(blockWidth, blockHeight, 1);
    const dim3 gridSize((width - 1) / blockWidth + 1, (height - 1) / blockHeight + 1, 1);

    gammaFunc<<<gridSize, blockSize>>>(dstimg, cn, width, height, numOfPixels, gammaValsPtr, howmany);
    CUDACHECK();

    CUDADBG(hipMemcpy(image.data, dstimg, numOfElems * sizeof(unsigned char), hipMemcpyDeviceToHost), );
    dstimg = NULL;
    CUDADBG(hipFree(gammaValsPtr), );
}